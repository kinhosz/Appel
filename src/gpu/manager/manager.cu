#ifndef APPEL_GPU_DISABLED

#include <gpu/manager.h>
#include <gpu/pragma.h>
#include <gpu/types/triangle.h>
#include <gpu/kernel.h>
#include <gpu/types/ray.h>

Manager::Manager(int maxTriangles, int batchsize) {
    this->maxTriangles = maxTriangles;
    this->BATCHSIZE = batchsize;

    this->threadsperblock_x = 1;
    this->threadsperblock_y = 1024;

    this->bufferN = (maxTriangles + threadsperblock_y - 1) / threadsperblock_y;

    size_t size;

    CUDA_STATUS(hipMalloc((void**)&dvc_bufferN, sizeof(int)));

    CUDA_STATUS(hipMalloc((void**)&blocks_N, sizeof(int)));
    CUDA_STATUS(hipMalloc((void**)&triangles_N, sizeof(int)));

    CUDA_STATUS(hipMalloc((void**)&rays_N, sizeof(int)));

    size = maxTriangles * sizeof(GTriangle);
    CUDA_STATUS(hipMalloc((void**)&cache, size));

    size = BATCHSIZE * sizeof(GRay);
    CUDA_STATUS(hipMalloc((void**)&dvc_rays, size));
    host_rays = (GRay*)malloc(size);

    size = (BATCHSIZE * bufferN) * sizeof(int);
    CUDA_STATUS(hipMalloc((void**)&buffer_idx, size));

    size = (BATCHSIZE * bufferN) * sizeof(float);
    CUDA_STATUS(hipMalloc((void**)&buffer_dist, size));

    size = BATCHSIZE * sizeof(int);
    CUDA_STATUS(hipMalloc((void**)&dvc_res_idx, size));
    host_res_idx = (int*)malloc(size);

    CUDA_STATUS(hipDeviceSynchronize());

    CUDA_STATUS(hipMemcpy(dvc_bufferN, &bufferN, sizeof(int),
        hipMemcpyHostToDevice));

    CUDA_STATUS(hipMemcpy(triangles_N, &maxTriangles, sizeof(int), 
        hipMemcpyHostToDevice));

    CUDA_STATUS(hipDeviceSynchronize());

    for(int i=0;i<maxTriangles;i++) {
        GTriangle gt;
        gt.host_id = -1;
        updateCache<<<1,1>>>(i, gt, cache);
        free_pos.push(i);
    }
}

Manager::~Manager() {
    CUDA_STATUS(hipFree(blocks_N));
    CUDA_STATUS(hipFree(triangles_N));
    CUDA_STATUS(hipFree(rays_N));
    CUDA_STATUS(hipFree(cache));
    CUDA_STATUS(hipFree(dvc_rays));
    free(host_rays);
    CUDA_STATUS(hipFree(dvc_res_idx));
    free(host_res_idx);

    CUDA_STATUS(hipFree(buffer_idx));
    CUDA_STATUS(hipFree(buffer_dist));
    CUDA_STATUS(hipFree(dvc_bufferN));
}

#endif
