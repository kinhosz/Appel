#ifndef APPEL_GPU_DISABLED

#include <gpu/manager.h>
#include <gpu/pragma.h>
#include <gpu/types/triangle.h>
#include <gpu/kernel.h>
#include <gpu/types/ray.h>

Manager::Manager(unsigned int maxTriangles) {
    this->maxTriangles = maxTriangles;
    this->threadsPerBlock = 1024;
    this->blocksPerGrid = (maxTriangles + threadsPerBlock - 1) / threadsPerBlock;
    this->free_pos = 0;

    size_t size = maxTriangles * sizeof(GTriangle);
    CUDA_STATUS(hipMalloc((void**)&cache, size));
    CUDA_STATUS(hipMallocManaged((void**)&result, sizeof(int)));
    CUDA_STATUS(hipMallocManaged((void**)&dvc_ray, sizeof(GRay)));
    CUDA_STATUS(hipMallocManaged((void**)&dvc_N, sizeof(int)));
    CUDA_STATUS(hipMallocManaged((void**)&dvc_BLOCK, sizeof(int)));
    
    size = blocksPerGrid * sizeof(float);
    CUDA_STATUS(hipMalloc((void**)&buffer_dist, size));
    size = blocksPerGrid * sizeof(int);
    CUDA_STATUS(hipMalloc((void**)&buffer_idx, size));

    size = maxTriangles * sizeof(GTriangle);
    tmp = (GTriangle* ) malloc(size);

    CUDA_STATUS(hipDeviceSynchronize());

    dvc_N[0] = (int)maxTriangles;
    dvc_BLOCK[0] = blocksPerGrid;

    for(int i=0;i<(int)maxTriangles;i++) {
        GTriangle t;
        t.host_id = -1;
        updateCache<<<1,1>>>(i, t, cache);
    }

    CUDA_STATUS(hipDeviceSynchronize());
}

Manager::~Manager() {
    CUDA_STATUS(hipFree(result));
    CUDA_STATUS(hipFree(cache));
    CUDA_STATUS(hipFree(dvc_N));
    CUDA_STATUS(hipFree(dvc_BLOCK));
    CUDA_STATUS(hipFree(dvc_ray));
    CUDA_STATUS(hipFree(buffer_dist));
    CUDA_STATUS(hipFree(buffer_idx));
    free(tmp);
}

#endif
